#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <vector>
#include <random>
#include <iostream>
#include <chrono>
#include <sstream>
#include <algorithm> // Required for std::sort

// Circle structure
struct Circle {
    float x, y, radius;
    float r, g, b, a; // Color components
    Circle(float x, float y, float radius, float r, float g, float b, float a)
        : x(x), y(y), radius(radius), r(r), g(g), b(b), a(a) {}
};

// Kernel to render circles in parallel
__global__ void renderCircles(Circle* d_circles, int num_circles, int canvas_width, int canvas_height, unsigned char* d_pixels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_circles) return;

    Circle circle = d_circles[idx];
    int centerX = static_cast<int>(circle.x);
    int centerY = static_cast<int>(circle.y);
    int radius = static_cast<int>(circle.radius);

    for (int dy = -radius; dy <= radius; ++dy) {
        for (int dx = -radius; dx <= radius; ++dx) {
            int x = centerX + dx;
            int y = centerY + dy;
            if (x < 0 || x >= canvas_width || y < 0 || y >= canvas_height) continue;

            if (dx * dx + dy * dy <= radius * radius) {
                int offset = (y * canvas_width + x) * 4;  // 4 for RGBA

                // Get current pixel color (destination)
                unsigned char dest_r = d_pixels[offset];
                unsigned char dest_g = d_pixels[offset + 1];
                unsigned char dest_b = d_pixels[offset + 2];
                unsigned char dest_a = d_pixels[offset + 3];

                // Calculate source color from circle's color and alpha
                unsigned char src_r = static_cast<unsigned char>(circle.r * 255);
                unsigned char src_g = static_cast<unsigned char>(circle.g * 255);
                unsigned char src_b = static_cast<unsigned char>(circle.b * 255);
                unsigned char src_a = static_cast<unsigned char>(circle.a * 255);

                // Perform alpha blending: output = alpha * source + (1 - alpha) * destination
                float alpha = circle.a;  // Assuming circle.a is in [0, 1]

                d_pixels[offset]     = static_cast<unsigned char>(alpha * src_r + (1 - alpha) * dest_r);
                d_pixels[offset + 1] = static_cast<unsigned char>(alpha * src_g + (1 - alpha) * dest_g);
                d_pixels[offset + 2] = static_cast<unsigned char>(alpha * src_b + (1 - alpha) * dest_b);
                d_pixels[offset + 3] = static_cast<unsigned char>(alpha * src_a + (1 - alpha) * dest_a);
            }
        }
    }
}

// Generate random circles on CPU
std::vector<Circle> generateCircles(int num_circles, int width, int height) {
    std::vector<Circle> circles;
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis_radius(5, 50);
    std::uniform_real_distribution<> dis_x(0, width);
    std::uniform_real_distribution<> dis_y(0, height);
    std::uniform_real_distribution<> dis_color(0.0f, 1.0f);

    for (int i = 0; i < num_circles; ++i) {
        circles.emplace_back(Circle{
            static_cast<float>(dis_x(gen)),
            static_cast<float>(dis_y(gen)),
            static_cast<float>(dis_radius(gen)),
            static_cast<float>(dis_color(gen)),
            static_cast<float>(dis_color(gen)),
            static_cast<float>(dis_color(gen)),
            static_cast<float>(dis_color(gen)),
        });
    }
    return circles;
}

int main() {
    const int canvas_width = 800;
    const int canvas_height = 600;

    // Inizializza il device CUDA
    hipError_t err;
    err = hipSetDevice(0);
    if (err != hipSuccess) {
        std::cerr << "Errore nel settare il device CUDA: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    int deviceCount = 0;
    err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess || deviceCount == 0) {
        std::cerr << "Nessun device CUDA trovato: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    std::cout << "Device count: " << deviceCount << std::endl;
    
    // Ottieni il massimo numero di thread per blocco supportato dalla GPU
    int max_threads_per_block;
    err = hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock, 0);
    if (err != hipSuccess) {
        std::cerr << "Errore nel recuperare l'attributo del device: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    std::cout << "Max threads per block: " << max_threads_per_block << std::endl;

    // Lista di diverse configurazioni di cerchi da testare
    std::vector<int> num_circles_config = {10, 100, 1000, 10000, 100000, 1000000};

    // Definisci configurazioni comuni per i thread per blocco
    std::vector<int> thread_configs = {256, 512, 1024};

    // Loop attraverso ogni configurazione di cerchi e thread
    for (int num_circles : num_circles_config) {
        // Genera i cerchi per la configurazione corrente
        std::vector<Circle> circles = generateCircles(num_circles, canvas_width, canvas_height);

        for (int threads_per_block : thread_configs) {
            // Salta configurazioni che superano il massimo supportato
            if (threads_per_block > max_threads_per_block)
                continue;

            // Allocazione della memoria sul device
            Circle* d_circles;
            unsigned char* d_pixels;
            hipMalloc(&d_circles, num_circles * sizeof(Circle));
            hipMalloc(&d_pixels, canvas_width * canvas_height * 4 * sizeof(unsigned char));
            hipMemset(d_pixels, 0, canvas_width * canvas_height * 4);

            // Copia i cerchi sulla GPU
            hipMemcpy(d_circles, circles.data(), num_circles * sizeof(Circle), hipMemcpyHostToDevice);

            // Calcola il numero di blocchi necessari
            int blocks = (num_circles + threads_per_block - 1) / threads_per_block;

            // Renderizza i cerchi sulla GPU
            auto start_time = std::chrono::high_resolution_clock::now();
            renderCircles<<<blocks, threads_per_block>>>(d_circles, num_circles, canvas_width, canvas_height, d_pixels);
            hipDeviceSynchronize();
            auto end_time = std::chrono::high_resolution_clock::now();

            // Copia i pixel dal device al host
            std::vector<unsigned char> pixels(canvas_width * canvas_height * 4);
            hipMemcpy(pixels.data(), d_pixels, canvas_width * canvas_height * 4, hipMemcpyDeviceToHost);

            // Salva l'immagine con un filename indicante il numero di cerchi e la configurazione di thread
            auto time_taken = std::chrono::duration<double>(end_time - start_time).count();
            std::stringstream filename;
            filename << "output/parallel_true_n_circles_" << num_circles
                     << "_threads_" << threads_per_block
                     << "_" << time_taken << ".png";
            std::cout << "Saving to: " << filename.str() << std::endl;
            stbi_write_png(filename.str().c_str(), canvas_width, canvas_height, 4, pixels.data(), canvas_width * 4);

            // Libera la memoria allocata sul device
            hipFree(d_circles);
            hipFree(d_pixels);

            std::cout << "Rendering completed with " << threads_per_block << " threads per block and " 
                      << num_circles << " circles in " << time_taken << " seconds.\n";
        }
    }

    return 0;
}
